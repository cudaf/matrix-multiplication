#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "_host.h"
#include "_simple.h"
//#include "_tiled.h"
#include "main.h"


int main() {
  int XR = 64;
  int XC = 64;
  int YC = 64;
  
  size_t A1 = XR * YC * sizeof(float);
  size_t X1 = XR * XC * sizeof(float);
  size_t Y1 = XC * YC * sizeof(float);

  float *a = (float*) malloc(A1);
  float *x = (float*) malloc(X1);
  float *y = (float*) malloc(Y1);
  
  float *exp = (float*) malloc(A1);
  test_populate(exp, x, y, XR, XC, YC);
  //PRINT2D(exp, 4, 4);

  printf("CPU matrix multiplication ...\n");
  test_print(exp, a, XR, YC, test_host(a, x, y, XR, XC, YC));
  //PRINT2D(a, 4, 4);

  printf("GPU matrix multiplication, simple ...\n");
  test_print(exp, a, XR, YC, test_simple(a, x, y, XR, XC, YC));
  //PRINT2D(a, 4, 4);

  //printf("CPU matrix multiplication, tiled ...\n");
  //test_print(exp, a, XR, YC, test_tiled(a, x, y, XR, XC, YC));
  return 0;
}
